#include <cstdio>
#include <cassert>

// #include "sixtracklib/sixtracklib.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// extern void run(double **indata, double **outdata, int npart );

__global__ void test( double* x, int npart )
{
    if( npart > 0 )
    {
        printf( "numbers : %.8f\r\n", x[ 0 ] );
        printf( "numbers : %.8f\r\n", x[ 1 ] );
        printf( "numbers : %.8f\r\n", x[ 2 ] );
        printf( "numbers : %.8f\r\n", x[ 3 ] );
        printf( "numbers : %.8f\r\n", x[ 4 ] );
        printf( "numbers : %.8f\r\n", x[ 5 ] );
        printf( "numbers : %.8f\r\n", x[ 6 ] );
    }

    return;
}

int main()
{
    int npart = 10;

    double* host_particle_buffer = 0;
    double* dev_particle_buffer  = 0;

    hipError_t err = hipSuccess;

    unsigned int device_flags = 0u;
    hipGetDeviceFlags( &device_flags );

    if( ( device_flags & hipDeviceMapHost ) != hipDeviceMapHost )
    {
        printf( "pinned memory not available with the "
                "cuda device -> aborting\r\n" );

        return 0;
    }

    err = hipHostAlloc( ( void** )&host_particle_buffer, npart * 240u,
                         hipHostMallocMapped );

    assert( err == hipSuccess );
    assert( host_particle_buffer != 0 );

    err = hipHostGetDevicePointer(
        ( void** )&dev_particle_buffer, host_particle_buffer, 0u );
    assert( err == hipSuccess );

    if( npart > 0 )
    {
        host_particle_buffer[  0 ] = 1.2345;
        host_particle_buffer[  1 ] = 2.2345;
        host_particle_buffer[  2 ] = 3.2345;
        host_particle_buffer[  3 ] = 4.2345;
        host_particle_buffer[  4 ] = 5.2345;
        host_particle_buffer[  5 ] = 6.2345;
        host_particle_buffer[  6 ] = 7.2345;
    }

    test<<< 1, 1 >>>( dev_particle_buffer, npart );


    err = hipHostFree( host_particle_buffer );
    host_particle_buffer = 0;
    assert( err == hipSuccess );

    return 0;
}

/* end: studies/study10/run_sample_fodo.c */
